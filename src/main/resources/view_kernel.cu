/*
 * Copyright (C) 2015 Laboratoire ThéMA - UMR 6049 - CNRS / Université de Franche-Comté
 * http://thema.univ-fcomte.fr
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_math_constants.h>

#define NCORE 512   

#define EARTH_DIAM 12740000

// TODO merge calcRay Bounds and Unbounded if the speed up is negligible

extern "C"
__global__ void calcRayDirect(int x0, int y0, float startZ, float destZ, float * dtm, int w, int h, 
        float res2D, int hasdsm, float * dsm, int earthCurv, float coefRefrac, unsigned char *view) {
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int x1, y1;
    if (tid < w) {
        x1 = tid;
        y1 = 0;
    } else if (tid < w + h) {
        x1 = w - 1;
        y1 = tid - w;
    } else if (tid < 2 * w + h) {
        x1 = tid - (w + h);
        y1 = h - 1;
    } else if (tid < 2 * w + 2 * h) {
        x1 = 0;
        y1 = tid - (2 * w + h);
    } else
        return;
    int ind = x0 + y0*w;
    const int ind1 = x1 + y1*w;
    const float z0 = dtm[ind] + startZ;
    const int dx = abs(x1 - x0);
    const int dy = abs(y1 - y0);
    const int sx = x0 < x1 ? 1 : -1;
    const int sy = y0 < y1 ? 1 : -1;
    int err = dx - dy;
    int xx = 0;
    int yy = 0;

    view[ind] = 1;
    
    float maxSlope = -1e127;
    float maxZ = -1e127;
    while (ind != ind1) {
        const int e2 = err * 2;
        if (e2 > -dy) {
            err -= dy;
            xx += sx;
            ind += sx;
        }
        if (e2 < dx) {
            err += dx;
            yy += sy;
            ind += sy*w;
        }
        
        float z = dtm[ind];    
        if(z == HIP_NAN_F) {
            return;
        }
        
        const float d2 = res2D*res2D * (xx*xx + yy*yy);
        
        if(earthCurv) {
            z -= (1 - coefRefrac) * d2 / EARTH_DIAM;
        }
        
        const float zSurf = z + (hasdsm ? dsm[ind] : 0);
        const float zView = destZ == -1 ? zSurf : (z + destZ);
        
        if (maxSlope >= 0 && zSurf <= maxZ && zView <= maxZ) {
            continue;
        }
        
        const float zzSurf = (zSurf - z0);
        const float slopeSurf = zzSurf * fabs(zzSurf) / d2;
        if(zView >= zSurf) {
            if(zView == zSurf && slopeSurf > maxSlope) {
                view[ind] = 1;
            } else {
                const double zzView = (zView - z0);
                const double slopeView = zzView*fabs(zzView) / d2;
                if(slopeView > maxSlope)
                    view[ind] = 1;
            }
        }
        if(slopeSurf > maxSlope) {
            maxSlope = slopeSurf;
        }
        if(zSurf > maxZ) {
            maxZ = zSurf;
        }
 
    }
}

extern "C"
__global__ void calcRayIndirect(int x0, int y0, float startZ, float destZ, float * dtm, int w, int h, 
        float res2D, int hasdsm, float * dsm, int earthCurv, float coefRefrac, unsigned char *view) {
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int x1, y1;
    if (tid < w) {
        x1 = tid;
        y1 = 0;
    } else if (tid < w + h) {
        x1 = w - 1;
        y1 = tid - w;
    } else if (tid < 2 * w + h) {
        x1 = tid - (w + h);
        y1 = h - 1;
    } else if (tid < 2 * w + 2 * h) {
        x1 = 0;
        y1 = tid - (2 * w + h);
    } else
        return;
    int ind = x0 + y0*w;
    const int ind1 = x1 + y1*w;
    
    const float dsmZ = hasdsm ? dsm[ind] : 0;
    if(destZ != -1 && destZ < dsmZ)
        return;
    const float z0 = dtm[ind] + (destZ != -1 ? destZ : dsmZ);
    const int dx = abs(x1 - x0);
    const int dy = abs(y1 - y0);
    const int sx = x0 < x1 ? 1 : -1;
    const int sy = y0 < y1 ? 1 : -1;
    int err = dx - dy;
    int xx = 0;
    int yy = 0;

    view[ind] = 1;
    
    float maxSlope = -1e127;
    float maxZ = -1e127;
    while (ind != ind1) {
        const int e2 = err * 2;
        if (e2 > -dy) {
            err -= dy;
            xx += sx;
            ind += sx;
        }
        if (e2 < dx) {
            err += dx;
            yy += sy;
            ind += sy*w;
        }
        float z = dtm[ind];
        if(z == HIP_NAN_F) {
            return;
        }
        const float dist = res2D * res2D * (xx * xx + yy * yy);
        if(earthCurv) {
            z -= (1 - coefRefrac) * dist / EARTH_DIAM;
        }
        if (maxSlope >= 0 && z + startZ <= maxZ) {
            continue;
        }
        float zz = (z + startZ - z0);
        float slope = zz * fabs(zz) / dist;
        if (slope > maxSlope) {
            view[ind] = 1;
        }
        const float ztot = z + (hasdsm ? dsm[ind] : 0);
        zz = ztot - z0;
        slope = zz * fabs(zz) / dist;
        if (slope > maxSlope)
            maxSlope = slope;
        if (ztot > maxZ)
            maxZ = ztot;
    }
}

extern "C"
__global__ void calcRayDirectBounded(int x0, int y0, float startZ, float destZ, float * dtm, int w, int h, 
            float res2D, int hasdsm, float * dsm, int earthCurv, float coefRefrac, unsigned char *view,
            // bounds
            float dMin2, float dMax2, float aleft, float aright, float sMin2, float sMax2) {
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int x1, y1;
    if (tid < w) {
        x1 = tid;
        y1 = 0;
    } else if (tid < w + h) {
        x1 = w - 1;
        y1 = tid - w;
    } else if (tid < 2 * w + h) {
        x1 = tid - (w + h);
        y1 = h - 1;
    } else if (tid < 2 * w + 2 * h) {
        x1 = 0;
        y1 = tid - (2 * w + h);
    } else
        return;
    
    double a = atan2((double)y0-y1, (double)x1-x0);
    if(a < 0)
        a += 2*M_PI;
    if(!(aright < aleft && a >= aright && a <= aleft || (aright >= aleft && (a >= aright || a <= aleft))))
        return;
        
    int ind = x0 + y0*w;
    const int ind1 = x1 + y1*w;
    float z0 = dtm[ind] + startZ;
    const int dx = abs(x1 - x0);
    const int dy = abs(y1 - y0);
    const int sx = x0 < x1 ? 1 : -1;
    const int sy = y0 < y1 ? 1 : -1;
    int err = dx - dy;
    int xx = 0;
    int yy = 0;

    if(sMin2 == -INFINITY && dMin2 == 0)
        view[ind] = 1;
    
    float maxSlope = sMin2;
    float maxZ = -1e127;
    while (ind != ind1) {
        const int e2 = err * 2;
        if (e2 > -dy) {
            err -= dy;
            xx += sx;
            ind += sx;
        }
        if (e2 < dx) {
            err += dx;
            yy += sy;
            ind += sy*w;
        }
        
        float z = dtm[ind];
        if(z == HIP_NAN_F) {
            return;
        }
        const float d2 = (res2D*res2D * (xx * xx + yy * yy));
        if(d2 >= dMax2) {
            return;
        }
        
        if(earthCurv) {
            z -= (1 - coefRefrac) * d2 / EARTH_DIAM;
        }
        
        const float zSurf = z + (hasdsm ? dsm[ind] : 0);
        const float zView = destZ == -1 ? zSurf : (z + destZ);
            
        if (maxSlope >= 0 && zSurf <= maxZ && zView <= maxZ) {
            continue;
        }
        const float zzSurf = (zSurf - z0);
        const float slopeSurf = zzSurf * fabs(zzSurf) / d2;
        if(slopeSurf > sMax2)
            return;
        if(d2 >= dMin2 && zView >= zSurf) {
            if(zView == zSurf && slopeSurf > maxSlope) {
                view[ind] = 1;
            } else {
                const double zzView = (zView - z0);
                const double slopeView = zzView*fabs(zzView) / d2;
                if(slopeView > maxSlope)
                    view[ind] = 1;
            }
        }
        
        if(slopeSurf > maxSlope) {
            maxSlope = slopeSurf;
        }
        if(zSurf > maxZ) {
            maxZ = zSurf;
        }
    }
}

extern "C"
__global__ void calcRayIndirectBounded(int x0, int y0, float startZ, float destZ, float * dtm, int w, int h, 
            float res2D, int hasdsm, float * dsm, int earthCurv, float coefRefrac, unsigned char *view,
            float dMin2, float dMax2, float aleft, float aright, float sMin2, float sMax2) {
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int x1, y1;
    if (tid < w) {
        x1 = tid;
        y1 = 0;
    } else if (tid < w + h) {
        x1 = w - 1;
        y1 = tid - w;
    } else if (tid < 2 * w + h) {
        x1 = tid - (w + h);
        y1 = h - 1;
    } else if (tid < 2 * w + 2 * h) {
        x1 = 0;
        y1 = tid - (2 * w + h);
    } else {
        return;
    }
    
    double a = atan2((double)y0-y1, (double)x1-x0);
    if(a < 0) {
        a += 2*M_PI;
    }
    if(!(aright < aleft && a >= aright && a <= aleft || (aright >= aleft && (a >= aright || a <= aleft)))) {
        return;
    }
    int ind = x0 + y0*w;
    const int ind1 = x1 + y1*w;
    
    const float dsmZ = hasdsm ? dsm[ind] : 0;
    if(destZ != -1 && destZ < dsmZ) {
        return;
    }
    const float z0 = dtm[ind] + (destZ != -1 ? destZ : dsmZ);
    const int dx = abs(x1 - x0);
    const int dy = abs(y1 - y0);
    const int sx = x0 < x1 ? 1 : -1;
    const int sy = y0 < y1 ? 1 : -1;
    int err = dx - dy;
    int xx = 0;
    int yy = 0;

    if(sMin2 == -INFINITY && dMin2 == 0) {
        view[ind] = 1;
    }
    
    float maxSlope = sMin2;
    float maxZ = -1e127;
    while (ind != ind1) {
        const int e2 = err * 2;
        if (e2 > -dy) {
            err -= dy;
            xx += sx;
            ind += sx;
        }
        if (e2 < dx) {
            err += dx;
            yy += sy;
            ind += sy*w;
        }
        float z = dtm[ind];
        if(z == HIP_NAN_F) {
            return;
        }
        const float d2 = res2D*res2D * (xx * xx + yy * yy);
        if(d2 >= dMax2) {
            return;
        }
        
        if(earthCurv) {
            z -= (1 - coefRefrac) * d2 / EARTH_DIAM;
        }
        if (maxSlope >= 0 && z + startZ <= maxZ) {
            continue;
        }
        
        float zz = (z + startZ - z0);
        const float slopeEye = zz * fabs(zz) / d2;
        if (slopeEye > maxSlope) {
            if(d2 >= dMin2 && slopeEye <= sMax2)
                view[ind] = 1;
        }
        const float ztot = z + (hasdsm ? dsm[ind] : 0);
        zz = ztot - z0;
        const float slope = zz * fabs(zz) / d2;
        if (slope > maxSlope) {
            maxSlope = slope;
        }
        if(maxSlope > sMax2) {
            return;
        }
        if (ztot > maxZ) {
            maxZ = ztot;
        }
    }
}

extern "C"
__global__  void calcRayTan(int x0, int y0, double startZ, float * dtm, int w, int h, 
            double res2D, int hasdsm, float * dsm, int earthCurv, float coefRefrac, int *view, int wa, double ares,
            double dMin, double dMax, double aleft, double aright, double sMin, double sMax) {
    
    const int ax = blockIdx.x * blockDim.x + threadIdx.x;
    if (ax >= wa)
        return;
    double a = (aleft - ax*ares);
    if(a < 0)
        a += 2*M_PI;

    if(!(aright < aleft && a >= aright && a <= aleft || (aright >= aleft && (a >= aright || a <= aleft))))
        return;

    int y1 = a >= 0 && a < M_PI ? 0 : h-1; // haut ou bas ?
    int x1 = a >= M_PI/2 && a < 1.5*M_PI ? 0 : w-1; // droite ou gauche ?
    int sens = x1 == 0 ? -1 : +1;
    
    int ddy = -round(tan(a) * abs(x1-x0));
    int y = y0 + sens * ddy;   
    if(y >= 0 && y < h) {
        y1 = y;   
    } else {
        int ddx = abs(round(tan(M_PI/2+a) * abs(y1-y0)));
        x1 = x0 + sens * ddx;
    }
    
/*
    if(a == 0) { // droite
        y1 = y0;
        x1 = w-1;
    } else if(a < M_PI/2) { // haut droit
        int dx = round(tan(M_PI/2-a) * y0); 
        if(x0+dx < w) {
            x1 = x0 + dx;
            y1 = 0;
        } else {
            int dy = round(tan(a) * (w-1-x0)); 
            x1 = w-1;
            y1 = y0-dy;
        }
    } else if(a == M_PI/2) { // haut 
        y1 = 0;
        x1 = x0;
    } else if(a < M_PI) { // haut gauche
        int dx = round(tan(a-M_PI/2) * y0); 
        if(x0-dx >= 0) {
            x1 = x0 - dx;
            y1 = 0;
        } else {
            int dy = round(tan(M_PI-a) * x0); 
            x1 = 0;
            y1 = y0-dy;
        }
    } else if(a == M_PI) { // gauche
        x1 = 0;
        y1 = y0;
    } else if(a < 1.5*M_PI) { // bas gauche
        int dx = round(tan(1.5*M_PI-a) * (h-1-y0)); 
        if(x0-dx >= 0) {
            x1 = x0 - dx;
            y1 = h-1;
        } else {
            int dy = round(tan(a-M_PI) * x0); 
            x1 = 0;
            y1 = y0+dy;
        }
    } else if(a == 1.5*M_PI) { // bas
        x1 = x0;
        y1 = h-1;
    } else { // bas droit
        int dx = round(tan(a-1.5*M_PI) * (h-1-y0)); 
        if(x0+dx >= 0 && x0+dx < w) {
            x1 = x0 + dx;
            y1 = h-1;
        } else {
            int dy = round(tan(2*M_PI-a) * (w-1-x0)); 
            x1 = w-1;
            y1 = y0+dy;
        }
    }
*/
    
    if(x1 < 0 || x1 >= w || y1 < 0 || y1 >= h) {
        for(int yz = 0; yz < wa/2; yz++) {
            view[yz*wa + ax] = -1;
        }
        return;
    }
    
    const int dx = abs(x1-x0);
    const int dy = abs(y1-y0);
    const int sx = x0 < x1 ? 1 : -1;
    const int sy = y0 < y1 ? 1 : -1;
    int err = dx-dy;
    int xx = 0;
    int yy = 0;
    int ind = x0 + y0*w;
    const int ind1 = x1 + y1*w;
    const double z0 = dtm[ind] + startZ;
    
    if(dMin == 0) {
        const double si = min(-startZ / (res2D/2), sMax);
        const int zi1 = (int) ((M_PI/2 - atan(si)) / ares);
        const int zi2 = (int) ((M_PI/2 - atan(sMin)) / ares);
        for(int yz = zi1; yz < zi2; yz++) {
            view[yz*wa + ax] = ind;
        }
    }
    double maxSlope = max(-startZ / (res2D/2), sMin);
    double maxZ = -1e127;
    while(ind != ind1) {
        const int e2 = err << 1;
        if(e2 > -dy) {
            err -= dy;
            xx += sx;
            ind += sx;
        }
        if(e2 < dx) {
            err += dx;
            yy += sy;
            ind += sy*w;
        }
        double z = dtm[ind] + (hasdsm ? dsm[ind] : 0);
        if(z == HIP_NAN) {
            return;
        }
        if(maxSlope >= 0 && z <= maxZ) {
            continue;
        }
        const double dist = res2D * sqrt((double)(xx*xx + yy*yy)) - copysign(1.0, z-z0)*res2D/2;
        if(dist > dMax) {
            return;
        }
        if(earthCurv) {
            z -= (1 - coefRefrac) * dist*dist / EARTH_DIAM;
        }
        const double slope = (z - z0) / dist;
        if(slope > maxSlope) {
            if(dist >= dMin) {
                const double s2 = min(sMax, slope);
                // tester Math.round à la place de ceil
                const int z2 = (int) round((M_PI/2 - atan(maxSlope)) / ares);
                const int z1 = (int) ((M_PI/2 - atan(s2)) / ares);

                for(int yz = z1; yz < z2; yz++) {
                    const int i = yz*wa + ax;
                    if(view[i] == -1) {
                        view[i] = (int) ind;
                    }
                }
            }   
            maxSlope = slope;
        }
        if(maxSlope > sMax) {
            return;
        }
        if(z > maxZ) {
            maxZ = z;
        }
    }

}

extern "C"
__global__ void sumView(unsigned char *g_idata, unsigned int n, int *g_odata) {
__shared__ int sdata[NCORE];
unsigned int tid = threadIdx.x;
unsigned int i = blockIdx.x*NCORE + tid;
unsigned int gridSize = NCORE*gridDim.x;
sdata[tid] = 0;

while (i < n) { sdata[tid] += g_idata[i]; i += gridSize; }
__syncthreads();
if (NCORE >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
if (NCORE >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
if (NCORE >= 128) { if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads(); }
if (tid < 32) {
    sdata[tid] += sdata[tid + 32];
     __syncthreads();
    sdata[tid] += sdata[tid + 16];
     __syncthreads();
    sdata[tid] += sdata[tid + 8];
     __syncthreads();
    sdata[tid] += sdata[tid + 4];
     __syncthreads();
    sdata[tid] += sdata[tid + 2];
     __syncthreads();
    sdata[tid] += sdata[tid + 1];
    }
if (tid == 0) 
    g_odata[blockIdx.x] = sdata[0];
}

extern "C"
__global__ void sumViewTan(int *g_idata, unsigned int n, int *g_odata) {
__shared__ int sdata[NCORE];
unsigned int tid = threadIdx.x;
unsigned int i = blockIdx.x*NCORE + tid;
unsigned int gridSize = NCORE*gridDim.x;
sdata[tid] = 0;

while (i < n) { 
    if(g_idata[i] > -1)
        sdata[tid]++; 
    i += gridSize; 
}
__syncthreads();
if (NCORE >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
if (NCORE >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
if (NCORE >= 128) { if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads(); }
if (tid < 32) {
    sdata[tid] += sdata[tid + 32];
     __syncthreads();
    sdata[tid] += sdata[tid + 16];
     __syncthreads();
    sdata[tid] += sdata[tid + 8];
     __syncthreads();
    sdata[tid] += sdata[tid + 4];
     __syncthreads();
    sdata[tid] += sdata[tid + 2];
     __syncthreads();
    sdata[tid] += sdata[tid + 1];
    }
if (tid == 0) 
    g_odata[blockIdx.x] = sdata[0];
}

extern "C"
__global__ void sumLandView(unsigned char * g_idata, unsigned int n, unsigned char * land, unsigned char code, int * g_odata) {
__shared__ int sdata[NCORE];
unsigned int tid = threadIdx.x;
unsigned int i = blockIdx.x*NCORE + tid;
unsigned int gridSize = NCORE*gridDim.x;
sdata[tid] = 0;

while (i < n) { 
    if(land[i] == code)
        sdata[tid] += g_idata[i]; 
    i += gridSize; 
}
__syncthreads();
if (NCORE >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
if (NCORE >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
if (NCORE >= 128) { if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads(); }
if (tid < 32) {
    sdata[tid] += sdata[tid + 32];
     __syncthreads();
    sdata[tid] += sdata[tid + 16];
     __syncthreads();
    sdata[tid] += sdata[tid + 8];
     __syncthreads();
    sdata[tid] += sdata[tid + 4];
     __syncthreads();
    sdata[tid] += sdata[tid + 2];
     __syncthreads();
    sdata[tid] += sdata[tid + 1];
    }
if (tid == 0) 
    g_odata[blockIdx.x] = sdata[0];
}

extern "C"
__global__ void sumLandViewTan(int * g_idata, unsigned int n, unsigned char * land, unsigned char code, int * g_odata) {
__shared__ int sdata[NCORE];
unsigned int tid = threadIdx.x;
unsigned int i = blockIdx.x*NCORE + tid;
unsigned int gridSize = NCORE*gridDim.x;
sdata[tid] = 0;

while (i < n) { 
    if(g_idata[i] > -1 && land[g_idata[i]] == code)
        sdata[tid]++; 
    i += gridSize; 
}
__syncthreads();
if (NCORE >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
if (NCORE >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
if (NCORE >= 128) { if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads(); }
if (tid < 32) {
    sdata[tid] += sdata[tid + 32];
     __syncthreads();
    sdata[tid] += sdata[tid + 16];
     __syncthreads();
    sdata[tid] += sdata[tid + 8];
     __syncthreads();
    sdata[tid] += sdata[tid + 4];
     __syncthreads();
    sdata[tid] += sdata[tid + 2];
     __syncthreads();
    sdata[tid] += sdata[tid + 1];
    }
if (tid == 0) 
    g_odata[blockIdx.x] = sdata[0];
}


extern "C"
__global__ void clearView(unsigned char *view, int size) {
    int tid = (blockIdx.x + gridDim.x * blockIdx.y) * blockDim.x + threadIdx.x;
    if (tid < size)
        view[tid] = 0;
}

extern "C"
__global__ void clearViewTan(int *view, int size) {
    int tid = (blockIdx.x + gridDim.x * blockIdx.y) * blockDim.x + threadIdx.x;
    if (tid < size)
        view[tid] = -1;
}

extern "C"
__global__ void addView(unsigned char *view, int * addView, int size) {
    int tid = (blockIdx.x + gridDim.x * blockIdx.y) * blockDim.x + threadIdx.x;
    if (tid < size)
        addView[tid] += view[tid];
}
